#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(float *a, float *b, float *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    float a[1] = {1.0e-45f};
    float b[1] = {1.0e-45f};
    float c[1] = {0.0f};

    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeof(float));
    hipMalloc((void **)&d_b, sizeof(float));
    hipMalloc((void **)&d_c, sizeof(float));

    hipMemcpy(d_a, a, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float), hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, sizeof(float), hipMemcpyDeviceToHost);

    printf("a = %e\n", a[0]);
    printf("b = %e\n", b[0]);
    printf("sum = %e\n", c[0]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
